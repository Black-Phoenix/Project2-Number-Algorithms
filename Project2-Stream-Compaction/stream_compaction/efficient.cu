#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
#include <stream_compaction/cpu.h>
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)


namespace StreamCompaction {
	#define block_size 256
	using StreamCompaction::Common::PerformanceTimer;
	
	namespace Efficient {
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}
		/*2 scan phases, see (link)[https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html] for more details*/
		__global__ void reduce_parallel(int n, int *data, int d) {
			int tmp_d = 1 << (d + 1);
			int index = (blockDim.x * blockIdx.x + threadIdx.x)*tmp_d;
			if (index >= n )
				return;
			data[index + tmp_d - 1] += data[index + (tmp_d>>1) - 1];
		}

		__global__ void downsweep_parallel(int n, int *data, int d) {
			int new_d = 1<<(d + 1);
			int index = (blockDim.x * blockIdx.x + threadIdx.x)*new_d;
			if (index >= n)
				return;
			int t = data[index + (new_d>>1) - 1];
			data[index + (new_d>>1) - 1] = data[index + new_d - 1];
			data[index + new_d - 1] += t;
		}
		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {
			timer().startGpuTimer();
			// allocate pointers to memory and copy data over
			int *dev_odata;
			int blocks = 0;
			int closest_pow2 = 1<<ilog2ceil(n);
			hipMalloc((void**)&dev_odata, closest_pow2 * sizeof(int));
			checkCUDAErrorWithLine("malloc failed!");
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorWithLine("memcpy failed!");
			// reduce phase
			// so we dont need to do the last round of computation because we zero it anyway
			for (int d = 0; d <= ilog2ceil(closest_pow2) - 2; d++) {
				// compute number of threads to spawn
				blocks = ceil((closest_pow2 / (1<<(d + 1)) + block_size - 1) / block_size);
				reduce_parallel <<<blocks, block_size >>> (closest_pow2, dev_odata, d);
				checkCUDAErrorWithLine("reduce phase failed!");
			}
			// down-sweep phase
			// zero last value
			hipMemset(dev_odata + (closest_pow2 - 1), 0, 1 * sizeof(int));
			for (int d = ceil(log2(closest_pow2) - 1); d >= 0; d--) {
				blocks = ceil((closest_pow2 / (1 << (d + 1)) + block_size - 1) / block_size);
				downsweep_parallel <<<blocks, block_size >>> (closest_pow2, dev_odata, d);
				checkCUDAErrorWithLine("downsweep phase failed!");
			}
			//read data back
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorWithLine("memcpy back failed!");
			timer().endGpuTimer();
		}

		/*Copy of scan but only works with cuda pointers*/
		void dev_scan(int n, int *dev_odata) {
			// allocate pointers to memory and copy data over
			int blocks = 0;
			int closest_pow2 = 1 << ilog2ceil(n);
			// reduce phase
			// so we dont need to do the last round of computation because we zero it anyway
			for (int d = 0; d <= ilog2ceil(closest_pow2) - 2; d++) {
				// compute number of threads to spawn
				blocks = (closest_pow2 / (1 << (d + 1)) + block_size - 1) / block_size;
				reduce_parallel << <blocks, block_size >> > (closest_pow2, dev_odata, d);
				checkCUDAErrorWithLine("reduce phase failed!");
			}
			// down-sweep phase
			// zero last value
			hipMemset(dev_odata + (closest_pow2 - 1), 0, 1 * sizeof(int));
			for (int d = ceil(log2(closest_pow2) - 1); d >= 0; d--) {
				blocks = (closest_pow2 / (1 << (d + 1)) + block_size - 1) / block_size;
				downsweep_parallel << <blocks, block_size >> > (closest_pow2, dev_odata, d);
				checkCUDAErrorWithLine("downsweep phase failed!");
			}
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int *odata, const int *idata) {
			timer().startGpuTimer();
			int closest_pow2 = 1<<ilog2ceil(n);
			int *dev_idata, *dev_odata, *dev_mask, *dev_indices;
			int blocks = ceil((closest_pow2 + block_size - 1) / block_size);
			hipMalloc((void**)&dev_idata, closest_pow2 * sizeof(int));
			hipMalloc((void**)&dev_odata, closest_pow2 * sizeof(int));
			hipMalloc((void**)&dev_mask, closest_pow2 * sizeof(int));
			hipMalloc((void**)&dev_indices, closest_pow2 * sizeof(int));
			// copy over idata
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorWithLine("mask gen failed!");
			Common::kernMapToBoolean <<<blocks, block_size >> > (n, dev_mask, dev_idata);
			checkCUDAErrorWithLine("mask gen failed!");
			// scan the mask array (can be done in parallel by using a balanced binary tree)
			hipMemcpy(dev_indices, dev_mask, closest_pow2 * sizeof(int), hipMemcpyDeviceToDevice);
			dev_scan(closest_pow2, dev_indices);
			checkCUDAErrorWithLine("dev scan failed!");
			// Scatter array (go to each position and copy the value)
			Common::kernScatter<<<blocks, block_size >> > (closest_pow2, dev_odata, dev_idata, dev_mask, dev_indices);

			checkCUDAErrorWithLine("scatter failed!");
			//read data back
			int res;
			hipMemcpy(&res, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			res = idata[n - 1] ? res + 1 : res;
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_mask);
			hipFree(dev_indices);
			timer().endGpuTimer();
			return res;
		}
	}
	namespace SharedMemory {
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}
		#define NUM_BANKS 16
		#define LOG_NUM_BANKS 4
		#define CONFLICT_FREE_OFFSET(n) \
			((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
		__global__ void dev_scan(int n, int *dev_odata, int *dev_idata, int *dev_block_sum)
		{
			/* Extriemly heavly based on https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
			   Main change is to use block_size instead of n and then put the sums of each block into a new array and scaning the array
			*/
			// Declare Share Memory
			__shared__ int temp[block_size + NUM_BANKS];
			int thid = threadIdx.x;
			int bid = blockIdx.x;
			int scan_offset = bid * block_size;
			int offset = 1; // to make this an exclusive scan
			int ai = thid<<1;
			int bi = ai + 1;
			int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
			int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
			temp[ai + bankOffsetA] = dev_idata[ai + scan_offset];
			temp[bi + bankOffsetB] = dev_idata[bi + scan_offset];
			for (int d = block_size >> 1; d > 0; d >>= 1)                    // build sum in place up the tree
			{
				__syncthreads();
				if (thid < d)
				{
					int ai = offset * ((thid<<1) + 1) - 1;
					int bi = offset * ((thid << 1) + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					temp[bi] += temp[ai];
				}
				offset<<=1;
			}
			__syncthreads();
			if (thid == 0) { 
				// place final sums of each block into extra array
				dev_block_sum[bid] = temp[block_size - 1 + CONFLICT_FREE_OFFSET(block_size - 1)];
				// zero last x cells because we are going to shift them when we do the downsweep
				temp[block_size - 1 + CONFLICT_FREE_OFFSET(block_size - 1)] = 0; 
			}
			// downsweep
			for (int d = 1; d < block_size; d<<=1) // traverse down tree & build scan
			{
				offset >>= 1;
				__syncthreads();
				if (thid < d)
				{
					int ai = offset * ((thid << 1) + 1) - 1;
					int bi = offset * ((thid << 1) + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					float t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}
			__syncthreads();
			dev_odata[ai + scan_offset] = temp[ai + bankOffsetA];
			dev_odata[bi + scan_offset] = temp[bi + bankOffsetB];
		}
		__global__ void add_offset(int n, int *data, int *dev_block_offset) {
			int bid = blockIdx.x;
			int index = blockDim.x * bid + threadIdx.x;
			if (index >= n)
				return;
			// add value to current section
			if (bid != 0) // to save a bunch of useless reads and write
				data[index] += dev_block_offset[bid];
		}
		void scan(int n, int *odata, int *idata) {
			timer().startGpuTimer();
			// allocate pointers to memory and copy data over
			int *dev_odata, *dev_idata, *dev_block_sum, *dev_block_offset;
			int *block_sum, *block_offset;
			int closest_pow2 = 1 << ilog2ceil(n);
			int blocks = ceil((closest_pow2 + block_size - 1) / block_size);
			// allocate buffers
			hipMalloc((void**)&dev_odata, closest_pow2 * sizeof(int));
			hipMalloc((void**)&dev_idata, closest_pow2 * sizeof(int));
			// allocate block global buffers
			hipMalloc((void**)&dev_block_sum, blocks * sizeof(int)); // each block gets 1 number to fill in 
			hipMalloc((void**)&dev_block_offset, blocks * sizeof(int));
			// cpu scan buffers
			block_sum = new int[blocks]();
			block_offset = new int[blocks]();
			checkCUDAErrorWithLine("malloc failed!");
			// copy over raw data
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorWithLine("memcpy failed!");
			// large prescan, pre alloc shared memory
			dev_scan<<<blocks, (block_size >> 1)>>>(closest_pow2, dev_odata, dev_idata, dev_block_sum);
			checkCUDAErrorWithLine("prescan fn failed!");
			// cpu scan the remaining blocks, because otherwise it could become recursive for large numbers
			hipMemcpy(block_sum, dev_block_sum, blocks * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorWithLine("memcpy to cpu failed!");
			StreamCompaction::CPU::scan(blocks, block_offset, block_sum);
			// copy data back over to cuda
			hipMemcpy(dev_block_offset, block_offset, blocks * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorWithLine("memcpy from cpu failed!");
			// add dev_block_offset to each block
			add_offset <<<blocks, block_size>>> (n, dev_odata, dev_block_offset);
			checkCUDAErrorWithLine("add_offset fn failed!");
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorWithLine("memcpy back failed!");
			// free memory
			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_block_sum);
			hipFree(dev_block_offset);
			delete block_sum;
			delete block_offset;
			timer().endGpuTimer();
		}
	}
}
